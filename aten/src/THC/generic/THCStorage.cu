
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCStorage.cu"
#else

void THCStorage_(fill)(THCState *state, THCStorage *self, real value)
{
  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<real> self_data(self->data);
  thrust::fill(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+self->size, value);
}

void THCStorage_(resize)(THCState *state, THCStorage *self, ptrdiff_t size)
{
  THArgCheck(size >= 0, 2, "invalid size");
  THAssert(self->allocator != NULL);
  int device;
  THCudaCheck(cudaGetDevice(&device));

  if(!(self->flag & TH_STORAGE_RESIZABLE))
    THError("Trying to resize storage that is not resizable");

  if (self->allocator->realloc) {
    cudaError_t err = (*self->allocator->realloc)(
      self->allocatorContext,
      (void**)&(self->data),
      self->size * sizeof(real),
      size * sizeof(real), THCState_getCurrentStream(state));
    if (err != cudaSuccess) {
      THCudaCheck(err);
    }
    self->size = size;
    self->device = device;
    return;
  }

  if(size == 0)
  {
    if(self->flag & TH_STORAGE_FREEMEM) {
      THCudaCheck(
        (*self->allocator->free)(self->allocatorContext, self->data));
    }
    self->data = NULL;
    self->size = 0;
    self->device = device;
  }
  else
  {
    real *data = NULL;
    cudaError_t err =
      (*self->allocator->malloc)(self->allocatorContext,
                                 (void**)&(data),
                                 size * sizeof(real),
                                 THCState_getCurrentStream(state));
    THCudaCheck(err);

    if (self->data) {
      // Enable p2p access when the memcpy is across devices
      THCState_getPeerToPeerAccess(state, device, self->device);

      THCudaCheck(cudaMemcpyAsync(data,
                                  self->data,
                                  THMin(self->size, size) * sizeof(real),
                                  cudaMemcpyDeviceToDevice,
                                  THCState_getCurrentStream(state)));
      if(self->flag & TH_STORAGE_FREEMEM) {
        THCudaCheck(
          (*self->allocator->free)(self->allocatorContext, self->data));
      }
    }

    self->data = data;
    self->size = size;
    self->device = device;
  }
}

THC_API int THCStorage_(getDevice)(THCState* state, const THCStorage* storage) {
  return storage->device;
}

#endif
